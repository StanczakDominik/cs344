
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_out, float *d_in)
{
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f;
}

int main (int argc, char ** argv)
{
  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  //generate input array on host
  float h_in[ARRAY_SIZE];
  for (int i = 0; i< ARRAY_SIZE; i++)
  {
    h_in[i] = float(i);
    printf("%f\n", h_in[i]);
  }

  float h_out[ARRAY_SIZE];

  //declare gpu memory pointers
  //just like cpu pointers
  float *d_in;
  float *d_out;

  //allocate gpu memory
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  //transfer array to device)
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);


  square<<<1, ARRAY_SIZE>>>(d_out, d_in); //cuda launch operator <<< >>>
  //launch one block on 64 threads (64 elements in array, 1:1



  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);


  for (int i =0; i<ARRAY_SIZE; i++)
  {
    printf("%f", h_out[i]);
    //okay this is a badass trick
    printf(((i % 4) != 3) ? "\t" : "\n");
    // printf(((i % 11) != 10) ? "\t" : "\n");
  }

  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
